#include <iostream>
#include <fstream>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>

// Define the REAL type as float
#define REAL float

#define REPEAT 1

#include "sphere_random_3d_points.cuh"
#include "normalization.cuh"

int main(int argc, char* argv[]) {
    
    
    if (argc != 
        #ifdef SAVE_OFF 
        6
        #else
        5
        #endif
    ) {
        std::cerr << "Usage: " << argv[0] << " [gpu/omp/seq] [n_points] [prob] [seed]"
        #ifdef SAVE_OFF
        "[output_name]"
        #endif
        << std::endl;
        return 1;
    }

    std::string mode(argv[1]);
    int n = std::stoi(argv[2]);
    double prob = std::stod(argv[3]);
    unsigned long seed = std::stoul(argv[4]);
    #ifdef SAVE_OFF
    std::string output_name(argv[5]);
    #endif


    REAL *x, *y, *z, *d_x, *d_y, *d_z;

    if (mode == "gpu") {
        #ifdef USE_GPU
        // Allocate memory on GPU
        checkCudaError(hipMalloc(&d_x, n * sizeof(REAL)), "hipMalloc d_x failed");
        checkCudaError(hipMalloc(&d_y, n * sizeof(REAL)), "hipMalloc d_y failed");
        checkCudaError(hipMalloc(&d_z, n * sizeof(REAL)), "hipMalloc d_z failed");
        #else
        std::cerr << "GPU mode not supported. Compile with -DUSE_GPU flag." << std::endl;
        return 1;
        #endif
        #ifdef SAVE_OFF
        // Allocate memory on CPU
        x = new REAL[n];
        y = new REAL[n];
        z = new REAL[n];
        #endif
    } else {
        // Allocate memory on CPU
        x = new REAL[n];
        y = new REAL[n];
        z = new REAL[n];
    }

    checkCudaError(hipDeviceSynchronize(), "CUDA Device Synchronization failed");
    auto start = std::chrono::high_resolution_clock::now();

    if (mode == "gpu") {
        #ifdef USE_GPU
        for (int i = 0; i < REPEAT; i++)
            generate_random_sphere_points_gpu<REAL>(n, d_x, d_y, d_z, prob, seed);

        #else
        std::cerr << "GPU mode not supported. Compile with -DUSE_GPU flag." << std::endl;
        return 1;
        #endif
    } else if (mode == "omp") {
        for (int i = 0; i < REPEAT; i++)
            generate_random_sphere_points_omp<REAL>(n, x, y, z, prob, seed);

        // Normalize CPU data in parallel
        normalize_cpu_data_omp(x, y, z, n);
    } else if (mode == "seq") {
        for (int i = 0; i < REPEAT; i++)
            generate_random_sphere_points<REAL>(n, x, y, z, prob, seed);

        // Normalize CPU data
        normalize_cpu_data(x, y, z, n);
    } else {
        std::cerr << "Invalid mode. Use 'gpu', 'omp', or 'seq'." << std::endl;
        return 1;
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time taken (" << mode << "): " << elapsed.count()/REPEAT << " seconds." << std::endl;

    #ifdef SAVE_OFF
    // Write to .off file
    std::string filename = output_name + ".off";
    std::ofstream offFile(filename);
    offFile << "OFF\n";
    offFile << n << " 0 0\n";
    if (mode == "gpu") {
        #ifdef USE_GPU
        // Allocate host memory for GPU data
        x = new REAL[n];
        y = new REAL[n];
        z = new REAL[n];
        // Copy from device to host
        checkCudaError(hipMemcpy(x, d_x, n * sizeof(REAL), hipMemcpyDeviceToHost), "hipMemcpy d_x to x failed");
        checkCudaError(hipMemcpy(y, d_y, n * sizeof(REAL), hipMemcpyDeviceToHost), "hipMemcpy d_y to y failed");
        checkCudaError(hipMemcpy(z, d_z, n * sizeof(REAL), hipMemcpyDeviceToHost), "hipMemcpy d_z to z failed");
        #endif
    }
    for (int i = 0; i < n; i++) {
        offFile << x[i] << " " << y[i] << " " << z[i] << "\n";
    }
    offFile.close();
    #endif

    if (mode == "gpu") {
        #ifdef USE_GPU
        // Free memory on GPU
        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        #endif        
        #ifdef SAVE_OFF
        // Free the host memory allocated for GPU data
        delete[] x;
        delete[] y;
        delete[] z;
        #endif
    } else {
        // Free allocated memory on CPU
        delete[] x;
        delete[] y;
        delete[] z;
    }

    return 0;
}
